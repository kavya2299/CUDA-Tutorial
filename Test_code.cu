#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define N  10000

__global__ void vector_add(float* a, float* b, float* out, int n) {
    int index = threadIdx.x;
    int stride = blockDim.x
        for (int i = 0; i < n; i++) {
            out[i] = a[i] + b[i];
        }
}

int main() {
    // declaring memory in host
    float* host_a, * b, * out;

    // declaring memory in device
    float* dev_a;

    // allocating memory on host
    host_a = (float*)malloc(sizeof(float) * N);
    b = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // initializing host_a and b
    for (int i = 0; i < N; i++) {
        host_a[i] = 0.1f;
        b[i] = 0.2f;
    }

    // allocating memory in device and copying host data  to device
    hipMalloc((void**)&dev_a, sizeof(float) * N);
    hipMemcpy(dev_a, host_a, sizeof(float) * N, hipMemcpyHostToDevice);
    vector_add << <1, 256 >> > (host_a, b, out, N);

    // deallocating memory from device
    hipFree(dev_a);

    // deallocating memory from host
    free(host_a);
    free(b);
    free(out);

    // returning success
    return 0;
}
